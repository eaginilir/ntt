#include <cstring>
#include <string>
#include <iostream>
#include <fstream>
#include <chrono>
#include <iomanip>
#include <sys/time.h>
#include <omp.h>
#include <algorithm>
#include <cmath>
#include <vector>
#include <hip/hip_runtime.h>


#define CUDA_CHECK(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << " - " << hipGetErrorString(error) << std::endl; \
            exit(1); \
        } \
    } while(0)


void fRead(int *a, int *b, int *n, int *p, int input_id)
{
    std::string strin = "./nttdata/" + std::to_string(input_id) + ".in";
    std::ifstream fin;
    fin.open(strin, std::ios::in);
    fin>>*n>>*p;
    for (int i = 0; i < *n; i++)
    {
        fin>>a[i];
    }
    for (int i = 0; i < *n; i++)
    {   
        fin>>b[i];
    }
}

void fCheck(int *ab, int n, int input_id)
{
    std::string strout = "./nttdata/" + std::to_string(input_id) + ".out";
    std::ifstream fin;
    fin.open(strout, std::ios::in);
    for (int i = 0; i < n * 2 - 1; i++)
    {
        int x;
        fin>>x;
        if(x != ab[i])
        {
            std::cout<<"多项式乘法结果错误"<<std::endl;
            return;
        }
    }
    std::cout<<"多项式乘法结果正确"<<std::endl;
    return;
}

void fWrite(int *ab, int n, int input_id)
{
    std::string strout = "./files/" + std::to_string(input_id) + ".out";
    std::ofstream fout;
    fout.open(strout, std::ios::out);
    for (int i = 0; i < n * 2 - 1; i++)
    {
        fout<<ab[i]<<'\n';
    }
}

// CPU快速幂
__host__ __device__ uint64_t power(uint64_t base, uint64_t exp, uint64_t mod) {
    uint64_t res = 1;
    base %= mod;
    while (exp > 0) {
        if (exp & 1) {
            res = ((__uint128_t)res * base) % mod;
        }
        base = ((__uint128_t)base * base) % mod;
        exp >>= 1;
    }
    return res;
}

// Barrett结构体
struct Barrett {
    uint64_t mod;
    uint64_t factor;
    __host__ __device__ Barrett() {}
    __host__ __device__ Barrett(uint64_t m) : mod(m) {
        factor = ~0ULL / m;
    }
    __host__ __device__ inline uint64_t reduce(uint64_t x) const {
        uint64_t q = ((uint64_t)(((__uint128_t)x * factor) >> 64));
        uint64_t r = x - q * mod;
        return r >= mod ? r - mod : r;
    }
    __host__ __device__ inline uint64_t mul(uint64_t a, uint64_t b) const {
        return reduce((__uint128_t)a * b);
    }
    __host__ __device__ inline uint64_t add(uint64_t a, uint64_t b) const {
        uint64_t r = a + b;
        return r >= mod ? r - mod : r;
    }
    __host__ __device__ inline uint64_t sub(uint64_t a, uint64_t b) const {
        return a >= b ? a - b : mod - (b - a);
    }
};

// GPU内核 - 位反转
__global__ void bit_reverse_kernel(uint64_t *a, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    int j = 0, t = idx;
    for (int bit = n >> 1; bit > 0; bit >>= 1) {
        j = (j << 1) | (t & 1);
        t >>= 1;
    }
    if (idx < j) {
        uint64_t tmp = a[idx];
        a[idx] = a[j];
        a[j] = tmp;
    }
}

// GPU内核 - NTT主循环
__global__ void ntt_kernel(uint64_t *a, int n, int len, const uint64_t *twiddles, Barrett barrett, int twiddle_offset) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_butterflies = n / len;
    if (idx >= total_butterflies * (len / 2)) return;
    int butterfly_group = idx / (len / 2);
    int j = idx % (len / 2);
    int i = butterfly_group * len;
    uint64_t w = twiddles[twiddle_offset + j];
    uint64_t u = a[i + j];
    uint64_t v = barrett.mul(w, a[i + j + len / 2]);
    a[i + j] = barrett.add(u, v);
    a[i + j + len / 2] = barrett.sub(u, v);
}

// GPU内核 - 逆NTT的最终除法
__global__ void inv_ntt_final_kernel(uint64_t *a, int n, uint64_t inv_n, Barrett barrett) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    a[idx] = barrett.mul(a[idx], inv_n);
}

// GPU内核 - 向量乘法
__global__ void pointwise_mul_kernel(uint64_t *c, const uint64_t *a, const uint64_t *b, int n, Barrett barrett) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    c[idx] = barrett.mul(a[idx], b[idx]);
}

class CudaBarrettNTT {
private:
    Barrett barrett;
    uint64_t *d_twiddles;
    int max_n;
    uint64_t p;
public:
    CudaBarrettNTT(uint64_t p_, int max_n_) : p(p_), max_n(max_n_) {
        barrett = Barrett(p_);
        d_twiddles = nullptr;
        precompute_twiddles();
    }
    ~CudaBarrettNTT() {
        if (d_twiddles) {
            CUDA_CHECK(hipFree(d_twiddles));
        }
    }
private:
    void precompute_twiddles() {
        int g = 3;
        std::vector<uint64_t> twiddles;
        // 正向
        for (int len = 2; len <= max_n; len <<= 1) {
            uint64_t wn = power(g, (p - 1) / len, p);
            uint64_t w = 1;
            for (int j = 0; j < len / 2; ++j) {
                twiddles.push_back(w);
                w = (__uint128_t(w) * wn) % p;
            }
        }
        // 逆向
        for (int len = 2; len <= max_n; len <<= 1) {
            uint64_t wn = power(g, (p - 1) / len, p);
            wn = power(wn, p - 2, p);
            uint64_t w = 1;
            for (int j = 0; j < len / 2; ++j) {
                twiddles.push_back(w);
                w = (__uint128_t(w) * wn) % p;
            }
        }
        CUDA_CHECK(hipMalloc(&d_twiddles, twiddles.size() * sizeof(uint64_t)));
        CUDA_CHECK(hipMemcpy(d_twiddles, twiddles.data(), twiddles.size() * sizeof(uint64_t), hipMemcpyHostToDevice));
    }
    int get_twiddle_offset(int len, bool inverse) {
        int offset = 0;
        for (int l = 2; l < len; l <<= 1) {
            offset += l / 2;
        }
        if (inverse) {
            int forward_total = 0;
            for (int l = 2; l <= max_n; l <<= 1) {
                forward_total += l / 2;
            }
            offset += forward_total;
        }
        return offset;
    }
public:
    void ntt_gpu(uint64_t *d_a, int n, bool inverse) {
        hipDeviceProp_t prop;
        CUDA_CHECK(hipGetDeviceProperties(&prop, 0));
        int blockSize = std::min(1024, prop.maxThreadsPerBlock);
        int gridSize;

        // 位反转
        gridSize = (n + blockSize - 1) / blockSize;
        bit_reverse_kernel<<<gridSize, blockSize>>>(d_a, n);
        CUDA_CHECK(hipDeviceSynchronize());

        // NTT主循环
        for (int len = 2; len <= n; len <<= 1) {
            int total_ops = n / len * (len / 2);
            gridSize = (total_ops + blockSize - 1) / blockSize;
            int twiddle_offset = get_twiddle_offset(len, inverse);
            ntt_kernel<<<gridSize, blockSize>>>(d_a, n, len, d_twiddles, barrett, twiddle_offset);
            CUDA_CHECK(hipDeviceSynchronize());
        }

        // 逆NTT的最终除法
        if (inverse) {
            uint64_t inv_n = power(n, p - 2, p);
            gridSize = (n + blockSize - 1) / blockSize;
            inv_ntt_final_kernel<<<gridSize, blockSize>>>(d_a, n, inv_n, barrett);
            CUDA_CHECK(hipDeviceSynchronize());
        }
    }

    void polynomial_multiply(const std::vector<uint64_t> &a, const std::vector<uint64_t> &b,
                           std::vector<uint64_t> &result, int n) {
        int len = 1;
        while (len < 2 * n) len <<= 1;
        result.resize(len);
        uint64_t *d_a, *d_b, *d_c;
        CUDA_CHECK(hipMalloc(&d_a, len * sizeof(uint64_t)));
        CUDA_CHECK(hipMalloc(&d_b, len * sizeof(uint64_t)));
        CUDA_CHECK(hipMalloc(&d_c, len * sizeof(uint64_t)));
        CUDA_CHECK(hipMemset(d_a, 0, len * sizeof(uint64_t)));
        CUDA_CHECK(hipMemset(d_b, 0, len * sizeof(uint64_t)));
        CUDA_CHECK(hipMemcpy(d_a, a.data(), n * sizeof(uint64_t), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_b, b.data(), n * sizeof(uint64_t), hipMemcpyHostToDevice));
        hipDeviceProp_t prop;
        CUDA_CHECK(hipGetDeviceProperties(&prop, 0));
        int blockSize = std::min(1024, prop.maxThreadsPerBlock);
        int gridSize = (len + blockSize - 1) / blockSize;

        ntt_gpu(d_a, len, false);
        ntt_gpu(d_b, len, false);
        pointwise_mul_kernel<<<gridSize, blockSize>>>(d_c, d_a, d_b, len, barrett);
        CUDA_CHECK(hipDeviceSynchronize());
        ntt_gpu(d_c, len, true);
        CUDA_CHECK(hipMemcpy(result.data(), d_c, len * sizeof(uint64_t), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipFree(d_a));
        CUDA_CHECK(hipFree(d_b));
        CUDA_CHECK(hipFree(d_c));
    }
};

int a[300000], b[300000], ab[300000];

int main(int argc, char *argv[]) {
    int deviceCount;
    CUDA_CHECK(hipGetDeviceCount(&deviceCount));
    if (deviceCount == 0) {
        std::cerr << "No CUDA devices found!" << std::endl;
        return 1;
    }
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, 0));
    std::cout << "Using GPU: " << prop.name << std::endl;
    std::cout << "SM count: " << prop.multiProcessorCount << std::endl;
    std::cout << "Max threads per block: " << prop.maxThreadsPerBlock << std::endl;
    std::cout << "Max threads per SM: " << prop.maxThreadsPerMultiProcessor << std::endl;

    int test_begin = 0;
    int test_end = 4;
    for(int i = test_begin; i <= test_end; ++i) {
        long double ans = 0;
        int n_, p_;
        fRead(a, b, &n_, &p_, i);
        memset(ab, 0, sizeof(ab));
        std::cout << "Processing test case " << i << " with n=" << n_ << ", p=" << p_ << std::endl;
        int len = 1;
        while(len < 2 * n_) len <<= 1;
        CudaBarrettNTT* cuda_ntt = nullptr;
        try {
            cuda_ntt = new CudaBarrettNTT(p_, len);
        } catch (const std::exception& e) {
            std::cerr << "Failed to create CUDA Barrett NTT for p=" << p_ << ": " << e.what() << std::endl;
            continue;
        }
        int epochs = 50;
        for (int epoch = 0; epoch < epochs; ++epoch) {
            std::vector<uint64_t> a_vec(n_), b_vec(n_), result;
            for (int j = 0; j < n_; ++j) {
                a_vec[j] = a[j];
                b_vec[j] = b[j];
            }
            auto Start = std::chrono::high_resolution_clock::now();
            cuda_ntt->polynomial_multiply(a_vec, b_vec, result, n_);
            auto End = std::chrono::high_resolution_clock::now();
            for (int j = 0; j < 2 * n_ - 1; ++j) {
                ab[j] = result[j] % p_;
            }
            std::chrono::duration<double, std::ratio<1,1000>> elapsed = End - Start;
            ans += elapsed.count();
        }
        fCheck(ab, n_, i);
        std::cout << "GPU average latency for n = " << n_ << " p = " << p_ 
                  << " : " << double(ans / epochs) << " (us) " << std::endl;
        fWrite(ab, n_, i);
    }
    return 0;
}